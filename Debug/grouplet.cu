#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" __global__ void __hmpp_acc_region__x2fftoqf_loop1D_1(__int32_t n, __int32_t* a, __int32_t* b, __int32_t* c);

void __hmpp_acc_region__x2fftoqf_internal_1(__int32_t n,__int32_t*  a,__int32_t*  b,__int32_t*  c, __int32_t* __hmpp_addr__i)
{
	dim3 threadsPerBlock((n-1)/128+1,1);
	dim3 numBlocks(32,4);
	__hmpp_acc_region__x2fftoqf_loop1D_1<<<numBlocks,threadsPerBlock>>>((n),a,b,c);
}
