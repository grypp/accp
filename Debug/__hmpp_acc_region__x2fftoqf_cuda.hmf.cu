#include "hip/hip_runtime.h"

#include <stdio.h>

#ifndef __HIPCC__
#include <stdlib.h>
#include <math.h>

#include <hmpprt/Grouplet.h>
#include <hmpprt/HostTypes.h>
#include <hmpprt/Context.h>
#include <hmpprt/CUDAGrid.h>
#include <hmpprt/CUDAModule.h>
#include <hmpprt/DeviceManager.h>
#include <hmpperr/hmpperr.h>

#ifdef _WIN32
#  define CDLT_API __declspec(dllexport)
#else /* ! _WIN32 */
#  define CDLT_API
#endif /* _WIN32 */

#else // ! __HIPCC__

#include <hmpprt/HostTypes.h>
#include <hmpprt/CUDAIntrinsics.h>

extern __shared__ int64_t hmpp_sharedmem[];
#endif // __HIPCC__

#define HMPPCG_WARP_SIZE 1  // Usually it should be 32, but we issued a bug on 2.1 capabilities devices



#ifndef __HIPCC__

#else


#endif

# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  CDLT_API void __hmpp_acc_region__x2fftoqf(hmpprt::s32 n, hmpprt::s32* a, hmpprt::s32* b, hmpprt::s32* c, hmpprt::s32* __hmpp_addr__i)
;
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
CDLT_API void __hmpp_acc_region__x2fftoqf_internal_1(hmpprt::s32 n, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  a, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  b, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  c, hmpprt::s32* __hmpp_addr__i)
;
#endif // __HIPCC__




#ifndef __HIPCC__
static hmpprt::CUDAGrid * __hmpp_acc_region__x2fftoqf_loop1D_1 = 0;
#else

extern "C" __global__ void __hmpp_acc_region__x2fftoqf_loop1D_1(hmpprt::s32 n, hmpprt::s32* a, hmpprt::s32* b, hmpprt::s32* c)
{
 # 7 "<preprocessor>"
 hmpprt::s32 i_1;
 i_1 = (hmpprt::gr_atidf());
 if (i_1 > n-1)
 {
  goto __hmppcg_label_1;
 }
 # 36 "vecadd.c"
 c[i_1] = a[i_1]+b[i_1];
 __hmppcg_label_1:;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
CDLT_API void __hmpp_acc_region__x2fftoqf_internal_1(hmpprt::s32 n, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  a, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  b, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32>  c, hmpprt::s32* __hmpp_addr__i)
{
 # 7 "<preprocessor>"
 hmpprt::s32 i;
 if (0 < n)
 {
  if (n-1 >= 0)
  {
   hmpprt::CUDAGridCall __hmppcg_call;
   __hmppcg_call.setSizeX((n-1)/128+1);
   __hmppcg_call.setSizeY(1);
   __hmppcg_call.setBlockSizeX(32);
   __hmppcg_call.setBlockSizeY(4);
   __hmppcg_call.addLocalParameter((hmpprt::s32) (n), "n");
   __hmppcg_call.addLocalParameter(&a, 8, "a");
   __hmppcg_call.addLocalParameter(&b, 8, "b");
   __hmppcg_call.addLocalParameter(&c, 8, "c");
   __hmppcg_call.launch(__hmpp_acc_region__x2fftoqf_loop1D_1, hmpprt::Context::getInstance()->getCUDADevice());
  }
  ;
  i = n;
 }
 else
 {
  i = 0;
 }
 # 16 "<preprocessor>"
 __hmpp_addr__i[0] = i;
}
#endif // __HIPCC__



# 5 "<preprocessor>"

#ifndef __HIPCC__
extern "C" CDLT_API  CDLT_API void __hmpp_acc_region__x2fftoqf(hmpprt::s32 n, hmpprt::s32* a, hmpprt::s32* b, hmpprt::s32* c, hmpprt::s32* __hmpp_addr__i)
{
 (__hmpp_acc_region__x2fftoqf_internal_1(n, hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (a), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (b), hmpprt::DevicePtr<hmpprt::MS_CUDA_GLOB,hmpprt::s32> (c), __hmpp_addr__i));
}
#endif // __HIPCC__




#ifndef __HIPCC__
extern "C" const char * hmpprt_cuda_get_gpu_code();

extern "C" CDLT_API void * hmpprt_init()
{
  try
  {
    hmpprt::CUDAModule * m = new hmpprt::CUDAModule(hmpprt_cuda_get_gpu_code());
    hmpprt::Context::getInstance()->getGrouplet()->setTarget(hmpprt::CUDA);
    __hmpp_acc_region__x2fftoqf_loop1D_1 = new hmpprt::CUDAGrid(m, "__hmpp_acc_region__x2fftoqf_loop1D_1");
    hmpprt::Context::getInstance()->getGrouplet()->addSignature("__hmpp_acc_region__x2fftoqf", "prototype __hmpp_acc_region__x2fftoqf(n: s32, a: ^cudaglob s32, b: ^cudaglob s32, c: ^cudaglob s32, __hmpp_addr__i: ^host s32)");

  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_init()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

#ifndef __HIPCC__
extern "C" CDLT_API void * hmpprt_fini()
{
  try
  {
    hmpprt::CUDAModule * m = 0;
    m = __hmpp_acc_region__x2fftoqf_loop1D_1->getModule();
    delete __hmpp_acc_region__x2fftoqf_loop1D_1;

    if(m)
      delete m;
  }
  catch (hmpperr::Error & e)
  {
    return e.clone();
  }
  catch(...)
  {
    fprintf(stderr,"Unexpected error in hmpprt_fini()\n");
    abort();
  }
  return 0;
}
#endif // __HIPCC__

// footer
