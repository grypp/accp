
#include <hip/hip_runtime.h>
extern __shared__ int64_t hmpp_sharedmem[];
extern "C" __global__ void __hmpp_acc_region__x2fftoqf_loop1D_1(__int32_t n, __int32_t* a, __int32_t* b, __int32_t* c)
{
 __int32_t i_1;
 i_1 = (blockDim.x * blockDim.y * blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x);
 if (i_1 > n-1)
 {
  goto __hmppcg_label_1;
 }
 c[i_1] = a[i_1]+b[i_1];
 __hmppcg_label_1:;
}

